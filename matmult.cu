
extern "C" {

#include <hip/hip_runtime.h>
#include <cblas.h>

void matmult_nat(int m,int n,int k,double *A,double *B,double *C);
void matmult_lib(int m,int n,int k,double *A,double *B,double *C);

void matmult_mkn(int m,int n,int k,double *A,double *B,double *C);
void matmult_mnk(int m,int n,int k,double *A,double *B,double *C);
void matmult_kmn(int m,int n,int k,double *A,double *B,double *C);
void matmult_knm(int m,int n,int k,double *A,double *B,double *C);
void matmult_nmk(int m,int n,int k,double *A,double *B,double *C);
void matmult_nkm(int m,int n,int k,double *A,double *B,double *C);

void matmult_blk(int m,int n,int k,double *A,double *B,double *C, int bs);
}

#define FOR_i_TO_m for (i = 0; i < m; i++)
#define FOR_j_TO_n for (j = 0; j < n; j++)
#define FOR_l_TO_k for (l = 0; l < k; l++)

#define RESET_C FOR_i_TO_m FOR_j_TO_n C[i * n + j] = 0;

#define MIN(a,b) ((a) < (b) ? a : b)

void matmult_nat(int m,int n,int k,double *A,double *B,double *C)
{
    int i, j, l;
    
    RESET_C

    FOR_i_TO_m
        FOR_j_TO_n
            FOR_l_TO_k
                C[i * n + j] += A[i * k + l] * B[l * n + j];
}

void matmult_lib(int m,int n,int k,double *A,double *B,double *C)
{
    cblas_dgemm(CblasRowMajor,CblasNoTrans,CblasNoTrans,m,n,k,1,A,k,B,n,0,C,n);
}

void matmult_mkn(int m,int n,int k,double *A,double *B,double *C)
{
    int i, j, l;
    
    RESET_C

    FOR_i_TO_m
        FOR_l_TO_k
            FOR_j_TO_n
                C[i * n + j] += A[i * k + l] * B[l * n + j];
}

void matmult_mnk(int m,int n,int k,double *A,double *B,double *C)
{
    matmult_nat(m, n, k, A, B, C);
}

void matmult_kmn(int m,int n,int k,double *A,double *B,double *C)
{    
    int i, j, l;
    
    RESET_C

    FOR_l_TO_k
        FOR_i_TO_m
            FOR_j_TO_n
                C[i * n + j] += A[i * k + l] * B[l * n + j];
}
void matmult_knm(int m,int n,int k,double *A,double *B,double *C)
{
    int i, j, l;
    
    RESET_C

    FOR_l_TO_k
        FOR_j_TO_n
            FOR_i_TO_m
                C[i * n + j] += A[i * k + l] * B[l * n + j];
}

void matmult_nmk(int m,int n,int k,double *A,double *B,double *C)
{
    int i, j, l;
    
    RESET_C

    FOR_j_TO_n
        FOR_i_TO_m
            FOR_l_TO_k
                C[i * n + j] += A[i * k + l] * B[l * n + j];
}

void matmult_nkm(int m,int n,int k,double *A,double *B,double *C)
{
    int i, j, l;
    
    RESET_C

    FOR_j_TO_n
        FOR_l_TO_k
            FOR_i_TO_m
                C[i * n + j] += A[i * k + l] * B[l * n + j];
}

void matmult_blk(int m,int n,int k,double *A,double *B,double *C, int bs)
{
    int I, J, L, i, j, l, limi, limj, liml;

    RESET_C

    for (I = 0; I < m; I+=bs)
	{
	limi = MIN(I+bs,m);
            for (L = 0; L < k; L+=bs)
	    	{
	   	liml = MIN(L+bs,k);
            	    for (J = 0; J < n; J+=bs)
			{
			limj = MIN(J+bs,n);
        		for (i = I; i < limi; i++)
            		    for (l = L; l < liml; l++)
                		for (j = J; j < limj; j++)
                            	    C[i * n + j] += A[i * k + l] * B[l * n + j];
			};
	     	};
		
	};
}
