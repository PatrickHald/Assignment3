#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

extern "C" {
#include <cblas.h>

#define FOR_i_TO_m for (i = 0; i < m; i++)
#define FOR_j_TO_n for (j = 0; j < n; j++)
#define FOR_l_TO_k for (l = 0; l < k; l++)

#define RESET_C FOR_i_TO_m FOR_j_TO_n C[i * n + j] = 0;

#define SIZE_A m*k*sizeof(double)
#define SIZE_B k*n*sizeof(double)
#define SIZE_C m*n*sizeof(double)

#define MIN(a,b) ((a) < (b) ? a : b)


void matmult_nat(int m,int n,int k,double *A,double *B,double *C);
void matmult_lib(int m,int n,int k,double *A,double *B,double *C);

void matmult_mkn(int m,int n,int k,double *A,double *B,double *C);
void matmult_mnk(int m,int n,int k,double *A,double *B,double *C);
void matmult_kmn(int m,int n,int k,double *A,double *B,double *C);
void matmult_knm(int m,int n,int k,double *A,double *B,double *C);
void matmult_nmk(int m,int n,int k,double *A,double *B,double *C);
void matmult_nkm(int m,int n,int k,double *A,double *B,double *C);

void matmult_blk(int m,int n,int k,double *A,double *B,double *C, int bs);

void matmult_gpu1(int m,int n,int k,double *A,double *B,double *C);
void matmult_gpu2(int m,int n,int k,double *A,double *B,double *C);
void matmult_gpu3(int m,int n,int k,double *A,double *B,double *C);
void matmult_gpu4(int m,int n,int k,double *A,double *B,double *C);
void matmult_gpu5(int m,int n,int k,double *A,double *B,double *C);
void matmult_gpu6(int m,int n,int k,double *A,double *B,double *C);

}

void matmult_nat(int m,int n,int k,double *A,double *B,double *C)
{
    int i, j, l;
    
    RESET_C

    FOR_i_TO_m
        FOR_j_TO_n
            FOR_l_TO_k
                C[i * n + j] += A[i * k + l] * B[l * n + j];
}

void matmult_lib(int m,int n,int k,double *A,double *B,double *C)
{
    cblas_dgemm(CblasRowMajor,CblasNoTrans,CblasNoTrans,m,n,k,1,A,k,B,n,0,C,n);
}

void matmult_mkn(int m,int n,int k,double *A,double *B,double *C)
{
    int i, j, l;
    
    RESET_C

    FOR_i_TO_m
        FOR_l_TO_k
            FOR_j_TO_n
                C[i * n + j] += A[i * k + l] * B[l * n + j];
}

void matmult_mnk(int m,int n,int k,double *A,double *B,double *C)
{
    matmult_nat(m, n, k, A, B, C);
}

void matmult_kmn(int m,int n,int k,double *A,double *B,double *C)
{    
    int i, j, l;
    
    RESET_C

    FOR_l_TO_k
        FOR_i_TO_m
            FOR_j_TO_n
                C[i * n + j] += A[i * k + l] * B[l * n + j];
}
void matmult_knm(int m,int n,int k,double *A,double *B,double *C)
{
    int i, j, l;
    
    RESET_C

    FOR_l_TO_k
        FOR_j_TO_n
            FOR_i_TO_m
                C[i * n + j] += A[i * k + l] * B[l * n + j];
}

void matmult_nmk(int m,int n,int k,double *A,double *B,double *C)
{
    int i, j, l;
    
    RESET_C

    FOR_j_TO_n
        FOR_i_TO_m
            FOR_l_TO_k
                C[i * n + j] += A[i * k + l] * B[l * n + j];
}

void matmult_nkm(int m,int n,int k,double *A,double *B,double *C)
{
    int i, j, l;
    
    RESET_C

    FOR_j_TO_n
        FOR_l_TO_k
            FOR_i_TO_m
                C[i * n + j] += A[i * k + l] * B[l * n + j];
}

void matmult_blk(int m,int n,int k,double *A,double *B,double *C, int bs)
{
    int I, J, L, i, j, l, limi, limj, liml;

    RESET_C

    for (I = 0; I < m; I+=bs)
	{
	limi = MIN(I+bs,m);
            for (L = 0; L < k; L+=bs)
	    	{
	   	liml = MIN(L+bs,k);
            	    for (J = 0; J < n; J+=bs)
			{
			limj = MIN(J+bs,n);
        		for (i = I; i < limi; i++)
            		    for (l = L; l < liml; l++)
                		for (j = J; j < limj; j++)
                            	    C[i * n + j] += A[i * k + l] * B[l * n + j];
			};
	     	};
		
	};
}

__global__ void gpu1(int m,int n,int k,double *A,double *B,double *C)
{
    int i, j, l;
    
    RESET_C

    FOR_i_TO_m
        FOR_l_TO_k
            FOR_j_TO_n
                atomicAdd(&C[i * n + j] , A[i * k + l] * B[l * n + j]);
}

void matmult_gpu1(int m,int n,int k,double *A,double *B,double *C)
{
    // The GPU uses only 1 thread

    double *d_A, *d_B, *d_C;
    
    // Allocate memory on the GPU
    hipMalloc((void**)&d_A, SIZE_A);
    hipMalloc((void**)&d_B, SIZE_B);
    hipMalloc((void**)&d_C, SIZE_C);

    // Transfer data from host to device 
    hipMemcpy(d_A, A, SIZE_A, hipMemcpyHostToDevice); 
    hipMemcpy(d_B, B, SIZE_B, hipMemcpyHostToDevice); 
    hipMemcpy(d_C, C, SIZE_C, hipMemcpyHostToDevice); 

    // Cuda launch
    gpu1<<<1,1>>>(m, n, k, d_A, d_B, d_C);
    hipDeviceSynchronize();

    // Transfer data from device to host 
    hipMemcpy(C, d_C, SIZE_C, hipMemcpyDeviceToHost); 

    // Free the allocated memory on the GPU
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
}

__global__ void gpu2(int m,int n,int k,double *A,double *B,double *C)
{
    int l;
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    double res = 0.0;
    if(i < m && j < n)
	FOR_l_TO_k
	    res += A[i * k + l] * B[l * n + j];
    if(i < m && j < n)
	C[i * n + j] = res;
    
}

void matmult_gpu2(int m,int n,int k,double *A,double *B,double *C)
{
    // We use one thread per element of C, which is m * n
   double *d_A, *d_B, *d_C;
    
    // Allocate memory on the GPU
    hipMalloc((void**)&d_A, SIZE_A);
    hipMalloc((void**)&d_B, SIZE_B);
    hipMalloc((void**)&d_C, SIZE_C);

    // Transfer data from host to device 
    hipMemcpy(d_A, A, SIZE_A, hipMemcpyHostToDevice); 
    hipMemcpy(d_B, B, SIZE_B, hipMemcpyHostToDevice); 
    hipMemcpy(d_C, C, SIZE_C, hipMemcpyHostToDevice); 

    // Cuda launch
    int K = 16;
    int G = SIZE_C / K + 1;
    dim3 dimGrid(G,G,1); // number of blocks 2D
    dim3 dimBlock(K,K,1); // number of threads per block 2D
    gpu2<<<dimGrid,dimBlock>>>(m, n, k, d_A, d_B, d_C);
    hipDeviceSynchronize();

    // Transfer data from device to host 
    hipMemcpy(C, d_C, SIZE_C, hipMemcpyDeviceToHost); 

    // Free the allocated memory on the GPU
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
 
}


void matmult_gpu3(int m,int n,int k,double *A,double *B,double *C)
{

}


void matmult_gpu4(int m,int n,int k,double *A,double *B,double *C)
{

}


void matmult_gpu5(int m,int n,int k,double *A,double *B,double *C)
{

}


void matmult_gpu6(int m,int n,int k,double *A,double *B,double *C)
{

}


