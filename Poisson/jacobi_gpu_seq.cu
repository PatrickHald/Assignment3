#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <stdlib.h>

__global__ void jacobi_gpu_seq(int N, double *array_in, double *array_out, double *fmatrix) {


//// SEQUENTIAL GPU KERNEL
    	for(int i = 1; i <= N; i++){ 
	    for(int j = 1; j <= N; j++){
	    	array_out[i*N+j] = 0.25*(array_in[(i-1)*N+j]+array_in[(i+1)*N+j]+array_in[i*N+j-1]+array_in[i*N+j+1]+fmatrix[i*N+j]);
		}
	   }
}
